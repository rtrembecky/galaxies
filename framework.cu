#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

// galaxy is stored as cartesian coordinates of its stars, each dimmension
// is in separate array
struct sGalaxy {
	float* x;
	float* y;
	float* z;
};

#include "kernel.cu"
#include "kernel_CPU.C"

#define N 2001
#define X 10

void generateGalaxies(sGalaxy A, sGalaxy B, int n) {
	for (int i = 0; i < n; i++) {
		// create star in A at random position first
		A.x[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		A.y[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		A.z[i] = 1000.0f * (float)rand() / (float)RAND_MAX;
		// create star in B near star A
		// in small probability, create more displaced star
		if ((float)rand() / (float)RAND_MAX < 0.01f) {
			B.x[i] = A.x[i] + 10.0f * (float)rand() / (float)RAND_MAX;
			B.y[i] = A.y[i] + 10.0f * (float)rand() / (float)RAND_MAX;
			B.z[i] = A.z[i] + 10.0f * (float)rand() / (float)RAND_MAX;
		}
		else {
			B.x[i] = A.x[i] + 1.0f * (float)rand() / (float)RAND_MAX;
			B.y[i] = A.y[i] + 1.0f * (float)rand() / (float)RAND_MAX;
			B.z[i] = A.z[i] + 1.0f * (float)rand() / (float)RAND_MAX;
		}
	}
}

int main(int argc, char **argv) {
	sGalaxy A, B;
	A.x = A.y = A.z = B.x = B.y = B.z = NULL;
	sGalaxy dA, dB;
	dA.x = dA.y = dA.z = dB.x = dB.y = dB.z = NULL;
	float diff_CPU, diff_GPU;

	// parse command line
	int device = 0;
	if (argc == 2)
		device = atoi(argv[1]);
	if (hipSetDevice(device) != hipSuccess) {
		fprintf(stderr, "Cannot set CUDA device!\n");
		exit(1);
	}
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, device);
	printf("Using device %d: \"%s\"\n", device, deviceProp.name);

	// create events for timing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate and set host memory
	A.x = (float*)malloc(N * sizeof(A.x[0]));
	A.y = (float*)malloc(N * sizeof(A.y[0]));
	A.z = (float*)malloc(N * sizeof(A.z[0]));
	B.x = (float*)malloc(N * sizeof(B.x[0]));
	B.y = (float*)malloc(N * sizeof(B.y[0]));
	B.z = (float*)malloc(N * sizeof(B.z[0]));
	generateGalaxies(A, B, N);

	// allocate and set device memory
	if (hipMalloc((void**)&dA.x, N * sizeof(dA.x[0])) != hipSuccess
		|| hipMalloc((void**)&dA.y, N * sizeof(dA.y[0])) != hipSuccess
		|| hipMalloc((void**)&dA.z, N * sizeof(dA.z[0])) != hipSuccess
		|| hipMalloc((void**)&dB.x, N * sizeof(dB.x[0])) != hipSuccess
		|| hipMalloc((void**)&dB.y, N * sizeof(dB.y[0])) != hipSuccess
		|| hipMalloc((void**)&dB.z, N * sizeof(dB.z[0])) != hipSuccess) {
		fprintf(stderr, "Device memory allocation error!\n");
		goto cleanup;
	}
	hipMemcpy(dA.x, A.x, N * sizeof(dA.x[0]), hipMemcpyHostToDevice);
	hipMemcpy(dA.y, A.y, N * sizeof(dA.y[0]), hipMemcpyHostToDevice);
	hipMemcpy(dA.z, A.z, N * sizeof(dA.z[0]), hipMemcpyHostToDevice);
	hipMemcpy(dB.x, B.x, N * sizeof(dB.x[0]), hipMemcpyHostToDevice);
	hipMemcpy(dB.y, B.y, N * sizeof(dB.y[0]), hipMemcpyHostToDevice);
	hipMemcpy(dB.z, B.z, N * sizeof(dB.z[0]), hipMemcpyHostToDevice);

	// solve on CPU
	printf("Solving on CPU...\n");
	hipEventRecord(start, 0);
	diff_CPU = solveCPU(A, B, N);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("CPU performance: %f megapairs/s\n",
		float(N)*float(N - 1) / 2.0f / time / 1e3f);

	// solve on GPU
	printf("Solving on GPU...\n");
	hipEventRecord(start, 0);
	// run it 10x for more accurately timing results
	for (int i = 0; i < X; i++)
		diff_GPU = solveGPU(dA, dB, N);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("GPU performance: %f megapairs/s\n",
		float(N)*float(N - 1) / 2.0f / time / 1e3f * X);

	printf("CPU diff: %f\nGPU diff: %f\n", diff_CPU, diff_GPU);
	// check GPU results
	if (fabsf((diff_CPU - diff_GPU) / ((diff_CPU + diff_GPU) / 2.0f)) < 0.01f)
		printf("Test OK :-).\n");
	else
		fprintf(stderr, "Data mismatch: %f should be %f :-(\n", diff_GPU, diff_CPU);

cleanup:
	hipEventDestroy(start);
	hipEventDestroy(stop);

	if (dA.x) hipFree(dA.x);
	if (dA.y) hipFree(dA.y);
	if (dA.z) hipFree(dA.z);
	if (dB.x) hipFree(dB.x);
	if (dB.y) hipFree(dB.y);
	if (dB.z) hipFree(dB.z);
	if (A.x) free(A.x);
	if (A.y) free(A.y);
	if (A.z) free(A.z);
	if (B.x) free(B.x);
	if (B.y) free(B.y);
	if (B.z) free(B.z);

	return 0;
}
